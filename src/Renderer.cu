#include "hip/hip_runtime.h"
#include "Renderer.cuh"
#include "View.hpp"

#include <SFML/Window.hpp>
#include <SFML/Graphics.hpp>
#include <SFML/OpenGL.hpp>
#include <cuda_gl_interop.h>
#include <numbers>

void cuda_check(hipError_t result)
{
	if (result == hipSuccess) return;
	throw std::exception(hipGetErrorString(result));
}

namespace man
{

Renderer::Renderer(const View& view) : view(view), sprite(new sf::Sprite{}), texture(new sf::Texture{})
{

}

__device__
float3 palette(float t, float3 a, float3 b, float3 c, float3 d)
{
	constexpr float Tau = std::numbers::pi * 2.0f;
	float x = a.x + b.x * cos(Tau * (c.x * t + d.x));
	float y = a.y + b.y * cos(Tau * (c.y * t + d.y));
	float z = a.z + b.z * cos(Tau * (c.z * t + d.z));
	return make_float3(x, y, z);
}

__device__
uint32_t convert_channel(float value)
{
	return min((uint32_t)(std::sqrt(value) * 256.0f), 255);
}

__global__
void kernel(Int2 resolution, Float2 view_center, Float2 view_extend, hipSurfaceObject_t surface)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= resolution.x || y >= resolution.y) return;

	Float half_resolution_x = (Float)resolution.x / (Float)2.0;
	Float half_resolution_y = (Float)resolution.y / (Float)2.0;

	Float normal_x = ((Float)x + (Float)0.5 - half_resolution_x) / half_resolution_x;
	Float normal_y = ((Float)y + (Float)0.5 - half_resolution_y) / half_resolution_y;

	Float first_position_x = fma(normal_x, view_extend.x, view_center.x);
	Float first_position_y = fma(normal_y, view_extend.y, view_center.y);

	auto julia_x = (Float)-0.05;
	auto julia_y = (Float)-0.66;

	Float position_x = first_position_x;
	Float position_y = first_position_y;

	uint32_t iteration = 0;

	while (iteration < 1024)
	{
		//		Float next_position_x = first_position_x + position_x * position_x - position_y * position_y;
		//		Float next_position_y = first_position_y + position_x * position_y * (Float)2.0;

		Float next_position_x = julia_x + position_x * position_x - position_y * position_y;
		Float next_position_y = julia_y + position_x * position_y * (Float)2.0;

		Float squared_magnitude = next_position_x * next_position_x + next_position_y * next_position_y;
		if (squared_magnitude >= 4.0f) break;

		position_x = next_position_x;
		position_y = next_position_y;

		++iteration;
	}

	float brightness = (float)iteration / 256.0f;
	brightness = brightness - truncf(brightness);
	uint32_t converted = min((uint32_t)(std::sqrt(brightness) * 256.0f), 255);
	uint32_t value = (converted << 16) | (converted << 8) | converted;

	//	float3 a = make_float3(0.5, 0.5, 0.5);
	//	float3 b = make_float3(0.5, 0.5, 0.5);
	//	float3 c = make_float3(4.0, 4.0, 4.0);
	//	float3 d = make_float3(0.00, 1.0f / 3.0f, 2.0f / 3.0f);
	//	float3 color = palette((float)iteration / 1024.0f, a, b, c, d);
	//	uint32_t value = (convert_channel(color.x) << 16) | (convert_channel(color.y) << 8) | convert_channel(color.z);

	surf2Dwrite<uint32_t>(0xFF000000 | value, surface, x * sizeof(uint32_t), y);
}

void Renderer::recreate_resources(Int2 resolution)
{
	if (graphics_resource != nullptr) cuda_check(hipGraphicsUnregisterResource(graphics_resource));

	texture->create(resolution.x, resolution.y);
	sprite->setTexture(*texture, true);

	cuda_check(hipGraphicsGLRegisterImage(
		&graphics_resource, texture->getNativeHandle(),
		GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
}

void Renderer::launch_kernels(Int2 resolution, Int2 block_size)
{
	hipResourceDesc description{ hipResourceTypeArray };

	cuda_check(hipGraphicsMapResources(1, &graphics_resource));
	cuda_check(hipGraphicsSubResourceGetMappedArray(&description.res.array.array, graphics_resource, 0, 0));

	hipSurfaceObject_t surface;
	cuda_check(hipCreateSurfaceObject(&surface, &description));

	Int2 block_count = (resolution + block_size - Int2(1)) / block_size;
	dim3 block_count_dim(block_count.x, block_count.y);
	dim3 block_size_dim(block_size.x, block_size.y);

	kernel<<<block_count_dim, block_size_dim>>>(resolution, view.get_center(), view.get_extend(), surface);

	cuda_check(hipDestroySurfaceObject(surface));
	cuda_check(hipGraphicsUnmapResources(1, &graphics_resource));
}

void Renderer::draw(sf::RenderWindow& window)
{
	Int2 resolution(window.getSize());

	if (resolution != Int2(texture->getSize())) recreate_resources(resolution);

	launch_kernels(resolution, { 16, 16 });
	cuda_check(hipDeviceSynchronize());

	window.draw(*sprite);
}

} // man
